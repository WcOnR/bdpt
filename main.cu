#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <fstream>
#include <math.h>
#include <stdio.h>
#include <hip/hip_vector_types.h>
#include "cutil_math.h"

using namespace std;

#define WIDTH 256
#define HEIGHT 256
#define ASPECT WIDTH/HEIGHT
#define SAMPLES 1
#define MAX_DEPTH 128
#define RPT 10
#define BLOCKS 512
#define THREADS 128

enum { NELEMS = WIDTH * HEIGHT };

typedef struct Ray {
	float3 direction;
	float3 origin;
}Ray;

typedef struct ray {
	float3 origin;
	float3 direction;
	float3 radiance;
	float terminate;
} ray;

typedef struct Camera {
	float3 lowleftcorner;
	float3 horizontal;
	float3 vertical;
	float3 origin;
}Camera;

typedef struct Material {
	float3 color;
	float reflectivity;
	float refractivity;
	float3 emissivity;
	float ior;
}Material;

typedef struct Photon {
	Ray reverse;
	float distance;
	int indx;
}Photon;

typedef struct Polygon {
	float3 dot[3];
	float3 normal;
	float D;
	Material material;
}Polygon;

typedef struct Sphere {
	float3 center;
	float radius;
	Material material;
	int pn;
}Sphere;

typedef struct Hit_record {
	float3 point;
	float3 normal;
	float depth;
	Material material;
}Hit_record;

//**************RANDOM**************//

__device__ float rand_f(hiprandState* globalState) {
	int indx = blockDim.x * blockIdx.x + threadIdx.x;
	hiprandState localState = globalState[indx];
	float RANDOM = hiprand_uniform( &localState );
	globalState[indx] = localState;
	return RANDOM;
}

__device__ float rand_f2m1(hiprandState * gState) {
	return rand_f(gState) * 2.0f - 1.0f;
}

//**************SUPPORTS_FUNCTIONS**************//

__device__ void create_camera(Camera &cam, float3 pov, float3 target, float fov) {
	float alpha = fov * M_PI / 180.0f;
	float vfov = tanf(alpha / 2.0f);
	float hfov = ASPECT * vfov;
	float3 dz = normalize(pov - target);
	float3 dx = normalize(cross(make_float3(0,1,0), dz));
	float3 dy = cross(dx, dz);
	cam.lowleftcorner = pov - hfov * dx - vfov * dy - dz;
	cam.horizontal = 2 * hfov * dx;
	cam.vertical = 2 * vfov * dy;
	cam.origin = pov;
}

__device__ Ray shoot_ray(int stage, Camera &cam, hiprandState* gState) {
	int indx = blockDim.x * blockIdx.x + threadIdx.x + stage * BLOCKS * THREADS;
	float w_off = (float)(indx % WIDTH + rand_f(gState)) / (float)(WIDTH);
	float h_off = (float)(indx / WIDTH + rand_f(gState)) / (float)(HEIGHT);
	Ray tmp;
	tmp.origin = cam.origin;
	tmp.direction = normalize(cam.lowleftcorner + w_off * cam.horizontal + h_off * cam.vertical - cam.origin);
	// tmp.direction = cam.lowleftcorner + w_off * cam.horizontal + h_off * cam.vertical - cam.origin;

	return tmp;
}

__device__ Ray shoot_ray(Sphere s, hiprandState *gState) {
	float3 dir = normalize(make_float3(rand_f2m1(gState), rand_f2m1(gState), rand_f2m1(gState)));
	return {dir, s.center + dir * s.radius};
}

__device__ float3 get_ray(Ray r, float k) {
	return r.origin + k * r.direction;
}

__device__ float3 reflection(float3 v1, float3 v2) {
	return v1 - 2.0f * dot(v1, v2) * v2;
}

__device__ bool diffusion(hiprandState* gState, Ray r, Ray &scattered, Hit_record &rec, float3 &attenuated){
	if (length(rec.material.emissivity) > 0) return false;
	float ran = rand_f(gState);

	if (ran < rec.material.reflectivity && rec.material.refractivity == 0.0f) {
		// metal
		float3 refl = normalize(reflection(r.direction, rec.normal));
		scattered = {refl, rec.point};
		attenuated = rec.material.color;
		return (dot(scattered.direction, rec.normal) > 0);
	} else if (rec.material.refractivity == 0.0f) {
		// matte
		float3 v1 = {rand_f2m1(gState), rand_f2m1(gState), rand_f2m1(gState)};
		v1 = normalize(v1);
		if (dot(v1, rec.normal) < 0.0f) v1 = -v1;
		scattered = {v1, rec.point};
		attenuated = rec.material.color;
		return true;
	}
	return true;
}

__device__ bool get_intersection(Ray r, Hit_record &rec, float dist, Sphere s) {
	float3 oc = r.origin - s.center;

	float a = dot(r.direction, r.direction);
	float b = 2.0f * dot(oc, r.direction);
	float c = dot(oc, oc) - s.radius * s.radius;
	float D = b * b - 4.0f * a * c;

	if (D > 0) {
		float d1 = (-b - sqrt(D)) / (2.0f * a);
		float d2 = (-b + sqrt(D)) / (2.0f * a);
		float root = (d1 < d2) ? d1 : d2;

		if (root < dist && root > 0.001f) {	
			rec.point = get_ray(r, root);
			rec.normal = (rec.point - s.center) / s.radius;
			rec.depth = root;
			rec.material = s.material;
			
			return true;
		}
	}
	return false;
}

__device__ bool nearest_intersection (Ray r, int &indx, Sphere *scene, Hit_record &rec, int n_obj) {
	float dist = 1E+37f;
	bool hit = false;
	for (int i = 0; i < n_obj; ++i) {
		if (get_intersection(r, rec, dist, scene[i])) {
			hit = true;
			dist = rec.depth;
			indx = i;
		}
	}
	return hit;

}

__device__ float3 inverse_raytrace(hiprandState* gState, Sphere *scene, Ray r, int n_obj) {
	Ray primary = r;
	Hit_record rec;
	float3 attenuated;
	float3 emitted;
	float3 composite = {0.0f, 0.0f, 0.0f};
	float3 counted = {1.0f, 1.0f, 1.0f};

	for (int i = 0; i < MAX_DEPTH; ++i) {
		int j;
		if (nearest_intersection(primary, j, scene, rec, n_obj)) {
			// return rec.material.color;
			emitted = rec.material.emissivity;
			Ray scattered;
			if (diffusion(gState, primary, scattered, rec, attenuated)) {
				primary = scattered;
				composite += (emitted + attenuated) * counted;
				counted = counted * attenuated;
			} else {
				return composite + emitted * counted;
			}
		} else {
			return {0, 0, 0};
		}
	}
	return {0, 0, 0};
	// return composite + emitted * counted;
}

__device__ Photon direct_raytrace(hiprandState* gState, Sphere *scene, Ray r, int n_obj) {
	Photon photon;
	Ray primary = r;
	Hit_record rec;
	int j;
	for (int i = 0; i < MAX_DEPTH; ++i) {
		if (nearest_intersection(primary, j, scene, rec, n_obj)) {
			if (rec.material.reflectivity == 0.0f) {
				photon.reverse.origin = rec.point;
				photon.distance = length(primary.origin - rec.point);
				photon.reverse.direction = -r.direction;
				photon.indx = j;
				return photon;
			} else if (rec.material.reflectivity > 0.0f) {
				float3 refl = normalize(reflection(primary.direction, rec.normal));
				primary = {refl, rec.point};
			} else if (length(rec.material.emissivity) > 0.0f) {
				photon = {{{0, 0, 0}, {0, 0, 0}}, 0, 2147483647};
				return photon;
			}
		} else {
			photon = {{{0, 0, 0}, {0, 0, 0}}, 0, 2147483647};
			return photon;
		}
	}
	photon = {{{0, 0, 0}, {0, 0, 0}}, 0, 2147483647};
	return photon;
}

//**************KERNEL**************//

__global__ void direct(hiprandState* gState, unsigned long seed, Sphere *scene, Photon *result, int n_obj, int *ligths, int light_obj) {
	int indx = blockDim.x * blockIdx.x + threadIdx.x;
	hiprand_init (seed, indx, 0, &gState[indx]);
	int j, shift = indx * RPT;
	Ray primary;
	for (int i = 0; i < RPT; ++i) {
		j = ligths[(int)(rand_f(gState) * light_obj)];
		primary = shoot_ray(scene[j], gState);
		result[shift + i] = direct_raytrace(gState, scene, primary, n_obj);
	}
}

__global__ void inverse(int stage, hiprandState* gState, Sphere *scene, float3 *result, int n, int n_obj) {
	int indx = blockDim.x * blockIdx.x + threadIdx.x;
	
	Camera cam;
	create_camera(cam, make_float3(-1, 100, 0), make_float3(0, 0, 0), 40.0f);
	
	if (indx < n) {
		Ray primary;
		result[indx + stage * BLOCKS * THREADS] = {0, 0, 0};
		float scaled = 1.0f / (float) SAMPLES;
		for (int i = 0; i < SAMPLES; ++i) {
			primary = shoot_ray(stage, cam, gState);
			result[indx + stage * BLOCKS * THREADS] += inverse_raytrace(gState, scene, primary, n_obj) * scaled;
		}
	}
}

//**************CPU**************//


// __host__ inline float gamma_correction(float val) {return val;}
// __host__ inline int gamma_correction(float val) {return val * 256;}
__host__ inline int gamma_correction(float val) { return int(255.0f * sqrt(fmaxf(0.0f, fminf(1.0f, val))));}

// __host__ void print_file_header(ofstream &file);
__host__ int compare (const void * a, const void * b);
__host__ void find_normal(Polygon &p);
__host__ Sphere * init_spheres(int &n_obj);
__host__ Polygon * init_polygons(int &n_obj);
__host__ void * alloc_mem_cpu(size_t size);
__host__ void mem_cpy_to_gpu(void *d, void *g, size_t size);
__host__ void * alloc_mem_gpu(size_t size);
__host__ void mem_cpy_to_cpu(void *g, void *d, size_t size);
__host__ void cudaErrors(hipError_t error);


int main() {
	/* Allocate vectors on host */
	int n_obj;
	size_t sizer = sizeof(float3) * NELEMS;
	size_t sizec = sizeof(hiprandState) * NELEMS;
	size_t sizep = sizeof(Photon) * NELEMS * RPT;
	Sphere *h_scene = NULL;
	Photon *h_photon = (Photon *) alloc_mem_cpu(sizep);
	// float3 *h_result = (float3 *) alloc_mem_cpu(sizer);

	h_scene = init_spheres(n_obj);
	size_t sizes = sizeof(Sphere) * n_obj;

	int light_obj = 0;
	for (int i = 0; i < n_obj; ++i) {
		if (length(h_scene[i].material.emissivity) > 0)
			++light_obj;
	}
	size_t sizel = sizeof(int *) * light_obj;
	int *h_lights = (int *) alloc_mem_cpu(sizel);
	for (int i = 0, j = 0; i < n_obj; ++i) {
		if (length(h_scene[i].material.emissivity) > 0) {
			h_lights[j] = i; 
			++j;
		}
	}
	/* Allocate vectors on device */
	Sphere *d_scene = NULL;
	int *d_ligths = NULL;
	Photon *d_photon = NULL;
	hiprandState* d_states = NULL;

	d_scene = (Sphere *)alloc_mem_gpu(sizes);	
	d_ligths = (int *) alloc_mem_gpu(sizel);
	d_photon = (Photon *) alloc_mem_gpu(sizep);
	// d_result = (float3 *)alloc_mem_gpu(sizer);    
	d_states = (hiprandState*)alloc_mem_gpu(sizec);

	/* Copy the host vectors to device */

	mem_cpy_to_gpu(h_scene, d_scene, sizes);	
	mem_cpy_to_gpu(h_lights, d_ligths, sizel);

	int threadsPerBlock = THREADS;
	int blocksPerGrid = BLOCKS; //(NELEMS + threadsPerBlock - 1) / threadsPerBlock;
	int maxStage = NELEMS/(BLOCKS * THREADS) + (NELEMS%(BLOCKS * THREADS) != 0) * 1;

	direct<<<blocksPerGrid, threadsPerBlock>>>(d_states, time(NULL), d_scene, d_photon, n_obj, d_ligths, light_obj);
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		cout << "Failed to launch Direct!\n";
		cudaErrors(error);
		exit(EXIT_FAILURE);
	}
	mem_cpy_to_cpu(d_photon, h_photon, sizep);
	cout << "direct method is done!\n";
	qsort(h_photon, NELEMS * RPT, sizeof(Photon), compare);
	int num_photon = 0, last_spher = 0;
	h_scene[0].pn = 0;
	// ofstream fi_out("out");
	for (int i = 0; i < NELEMS * RPT; ++i) {
		if (h_photon[i].distance != 0) {
			++num_photon;
			if (last_spher != h_photon[i].indx) {
				last_spher = h_photon[i].indx;
				h_scene[last_spher].pn = i;
			}
			// Ray r = h_photon[i].reverse;
			// fi_out << r.origin.x << ":" << r.origin.y << ":" << r.origin.z << "\t\t";
			// fi_out << r.direction.x << ":" << r.direction.y << ":" << r.direction.z << "\t\t";
			// fi_out << h_photon[i].distance << ":" << h_photon[i].indx << "\n";
		}
	}
	for (int i = 0; i < n_obj; ++i) {
		cout << h_scene[i].pn << endl;
	}
	// fi_out.close();
	// free(h_photon);
	// hipFree(d_photon);

	mem_cpy_to_gpu(h_scene, d_scene, sizes);
	float3 *h_result = (float3 *) alloc_mem_cpu(sizer);
	float3 *d_result = NULL;	
	d_result = (float3 *) alloc_mem_gpu(sizer); 
	for (int i = 0; i < maxStage; ++i) {
		inverse<<<blocksPerGrid, threadsPerBlock>>>(i, d_states, d_scene, d_result, BLOCKS * THREADS, n_obj);
		error = hipGetLastError();
		if (error != hipSuccess) {
			cout << "Failed to launch Inverse method!\n";
			cudaErrors(error);
			exit(EXIT_FAILURE);
		}
	}
	mem_cpy_to_cpu(d_result, h_result, sizer);
	cout << "inverse method is done!\n";
	ofstream fout("img.ppm");
	fout << "P3\n" << WIDTH << " " << HEIGHT << "\n255\n";
	for (int i = 0; i < NELEMS; ++i) {
		fout << gamma_correction(h_result[i].x) << "\t\t\t\t"
				<< gamma_correction(h_result[i].y) << "\t\t\t\t"
				<< gamma_correction(h_result[i].z) << "\n";
	}
	fout.close();
	cout << "We Fine!\n";
	hipFree(d_scene);
	hipFree(d_result);
	hipFree(d_states);
	free(h_result);
	free(h_scene);
	hipDeviceReset();
	return 0;
}

// __host__ void print_file_header(ofstream &file) {
// 	file << (uint8_t)'B' << (uint8_t)'M' << (uint32_t)WIDTH * HEIGHT * 3 + 54;
// 	file << (uint16_t)0 << (uint16_t)0 << (uint32_t)54 << (uint32_t)40;
// 	file << (uint32_t)WIDTH << (uint32_t)HEIGHT << (uint16_t)1;
// 	file << (uint16_t)24 << (uint32_t)0 << (uint32_t)0 << (uint32_t)0;
// 	file << (uint32_t)0 << (uint32_t)0 << (uint32_t)0;
// }

__host__ int compare(const void * a, const void * b) {
	Photon ap = *(Photon *) a, bp = *(Photon *) b;
	return ap.indx - bp.indx;
}

__host__ void find_normal(Polygon &p) {
	float3 a = p.dot[1] - p.dot[0], b = p.dot[2] - p.dot[1];
	p.normal = normalize(cross(a, b));
}

__host__ void find_factor(Polygon &p) {
	p.D = - p.normal.x * p.dot[0].x - p.normal.y * p.dot[0].y - p.normal.z * p.dot[0].z;
}

__host__ Material * init_materials(int &m) {
	ifstream inMat("inMat");
	inMat >> m;
	size_t sizeMat = sizeof(Material) * m;
	Material *materials = (Material *) alloc_mem_cpu(sizeMat);
    for (int i = 0; i < m; ++i) {
		inMat >> materials[i].color.x >> materials[i].color.y >> materials[i].color.z;
		inMat >> materials[i].reflectivity;
		inMat >> materials[i].refractivity;
		inMat >> materials[i].emissivity.x >> materials[i].emissivity.y >> materials[i].emissivity.z;
		inMat >> materials[i].ior;
	}
	inMat.close();
	return materials;
}

__host__ Sphere * init_spheres(int &n_obj) {
	int m;
	Material *materials = init_materials(m);
	ifstream infile("inSpher");
	infile >> n_obj;

	size_t sizeScene = sizeof(Sphere) * n_obj;
	Sphere *scene = (Sphere *)alloc_mem_cpu(sizeScene);

	for (int i = 0; i < n_obj; ++i) {
		infile >> scene[i].center.x >> scene[i].center.y >> scene[i].center.z;
		infile >> scene[i].radius;
		int tmp;
		infile >> tmp;
		scene[i].material = materials[tmp];
		scene[i].pn = -1;
	}
	infile.close();
	free(materials);
	return scene;
}

__host__ Polygon * init_polygons(int &n_obj) {
	int m;
	Material *materials = init_materials(m);
	ifstream infile("inPolygon");
	infile >> n_obj;

	size_t sizeScene = sizeof(Polygon) * n_obj;
	Polygon *scene = (Polygon *)alloc_mem_cpu(sizeScene);

	for (int i = 0; i < n_obj; ++i) {
		for (int j = 0; j < 3; ++j)
	    	infile >> scene[i].dot[j].x >> scene[i].dot[j].y >> scene[i].dot[j].z;
	    int tmp;
	    infile >> tmp;
	    scene[i].material = materials[tmp];
	    find_normal(scene[i]);
	    find_factor(scene[i]);
	    // cout << scene[i].normal.x << ":" << scene[i].normal.y << ":" << scene[i].normal.z << "\n\n";
	}
	infile.close();

	free(materials);
	return scene;
}

__host__ void * alloc_mem_cpu(size_t size) {
	void *ptr = malloc(size);
	if (ptr == NULL) {
		cout << "Allocation error.\n";
		exit(EXIT_FAILURE);
	}
	return ptr;
}

__host__ void mem_cpy_to_gpu(void *h, void *d, size_t size) {
	if (hipMemcpy(d, h, size, hipMemcpyHostToDevice) != hipSuccess) {
		cout << "Host to device copying failed!\n";
		exit(EXIT_FAILURE);
	}
}

__host__ void * alloc_mem_gpu(size_t size) {
	void *ptr; 
	if (hipMalloc ((void **)&ptr, size) != hipSuccess) {
		cout << "Host to device copying failed\n";
		exit(EXIT_FAILURE);
	}
	return ptr;
}

__host__ void mem_cpy_to_cpu(void *d, void *h, size_t size) {
	if (hipMemcpy(h, d, size, hipMemcpyDeviceToHost) != hipSuccess) {
		cout << "Device to host copying failed\n";
		exit(EXIT_FAILURE);
	}
}

__host__ void cudaErrors(hipError_t error) {
	switch(error) {
		case hipSuccess: cout << "Success!\n"; break;
		case hipErrorMissingConfiguration: cout << "hipErrorMissingConfiguration!\n"; break;
		case hipErrorOutOfMemory: cout << "hipErrorOutOfMemory!\n"; break;
		case hipErrorNotInitialized: cout << "hipErrorNotInitialized!\n"; break;
		case hipErrorLaunchFailure: cout << "hipErrorLaunchFailure!\n"; break;
		case hipErrorPriorLaunchFailure: cout << "hipErrorPriorLaunchFailure!\n"; break;
		case hipErrorLaunchTimeOut: cout << "hipErrorLaunchTimeOut!\n"; break;
		case hipErrorLaunchOutOfResources: cout << "hipErrorLaunchOutOfResources!\n"; break;
		case hipErrorInvalidDeviceFunction: cout << "hipErrorInvalidDeviceFunction!\n"; break;
		case hipErrorInvalidConfiguration: cout << "hipErrorInvalidConfiguration!\n"; break;
		case hipErrorInvalidDevice: cout << "hipErrorInvalidDevice!\n"; break;
		case hipErrorInvalidValue: cout << "hipErrorInvalidValue!\n"; break;
		case hipErrorInvalidPitchValue: cout << "hipErrorInvalidPitchValue!\n"; break;
		case hipErrorInvalidSymbol: cout << "hipErrorInvalidSymbol!\n"; break;
		case hipErrorMapFailed: cout << "hipErrorMapFailed!\n"; break;
		case hipErrorUnmapFailed: cout << "hipErrorUnmapFailed!\n"; break;
		case cudaErrorInvalidHostPointer: cout << "cudaErrorInvalidHostPointer!\n"; break;
		case hipErrorInvalidDevicePointer: cout << "hipErrorInvalidDevicePointer!\n"; break;
		case hipErrorInvalidTexture: cout << "hipErrorInvalidTexture!\n"; break;
		case cudaErrorInvalidTextureBinding: cout << "cudaErrorInvalidTextureBinding!\n"; break;
		case hipErrorInvalidChannelDescriptor: cout << "hipErrorInvalidChannelDescriptor!\n"; break;
		case hipErrorInvalidMemcpyDirection: cout << "hipErrorInvalidMemcpyDirection!\n"; break;
		case cudaErrorAddressOfConstant: cout << "cudaErrorAddressOfConstant!\n"; break;
		case cudaErrorTextureFetchFailed: cout << "cudaErrorTextureFetchFailed!\n"; break;
		case cudaErrorTextureNotBound: cout << "cudaErrorTextureNotBound!\n"; break;
		case cudaErrorSynchronizationError: cout << "cudaErrorSynchronizationError!\n"; break;
		case cudaErrorInvalidFilterSetting: cout << "cudaErrorInvalidFilterSetting!\n"; break;
		case cudaErrorInvalidNormSetting: cout << "cudaErrorInvalidNormSetting!\n"; break;
		case cudaErrorMixedDeviceExecution: cout << "cudaErrorMixedDeviceExecution!\n"; break;
		// case cudaErrorcudartUnloading: cout << "cudaErrorcudartUnloading!\n"; break;
		case hipErrorUnknown: cout << "hipErrorUnknown!\n"; break;
		case cudaErrorNotYetImplemented: cout << "cudaErrorNotYetImplemented!\n"; break;
		case cudaErrorMemoryValueTooLarge: cout << "cudaErrorMemoryValueTooLarge!\n"; break;
		case hipErrorInvalidHandle: cout << "hipErrorInvalidHandle!\n"; break;
		case hipErrorNotReady: cout << "hipErrorNotReady!\n"; break;
		case hipErrorInsufficientDriver: cout << "hipErrorInsufficientDriver!\n"; break;
		case hipErrorSetOnActiveProcess: cout << "hipErrorSetOnActiveProcess!\n"; break;
		case cudaErrorInvalidSurface: cout << "cudaErrorInvalidSurface!\n"; break;
		case hipErrorNoDevice: cout << "hipErrorNoDevice!\n"; break;
		case hipErrorECCNotCorrectable: cout << "hipErrorECCNotCorrectable!\n"; break;
		case hipErrorSharedObjectSymbolNotFound: cout << "hipErrorSharedObjectSymbolNotFound!\n"; break;
		case hipErrorSharedObjectInitFailed: cout << "hipErrorSharedObjectInitFailed!\n"; break;
		case hipErrorUnsupportedLimit: cout << "hipErrorUnsupportedLimit!\n"; break;
		case cudaErrorDuplicateVariableName: cout << "cudaErrorDuplicateVariableName!\n"; break;
		case cudaErrorDuplicateTextureName: cout << "cudaErrorDuplicateTextureName!\n"; break;
		case cudaErrorDuplicateSurfaceName: cout << "cudaErrorDuplicateSurfaceName!\n"; break;
		case cudaErrorDevicesUnavailable: cout << "cudaErrorDevicesUnavailable!\n"; break;
		case hipErrorInvalidImage: cout << "hipErrorInvalidImage!\n"; break;
		case hipErrorNoBinaryForGpu: cout << "hipErrorNoBinaryForGpu!\n"; break;
		case cudaErrorIncompatibleDriverContext: cout << "cudaErrorIncompatibleDriverContext!\n"; break;
		case cudaErrorStartupFailure: cout << "cudaErrorStartupFailure!\n"; break;
		case cudaErrorApiFailureBase: cout << "cudaErrorApiFailureBase!\n"; break;
	}
}
		
